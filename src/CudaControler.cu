#include "hip/hip_runtime.h"
//MIT License
//Copyright (c) 2019 Gonzalo G Campos



#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <Windows.h>

#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>


#include <CudaControler.h>
#include <Values.h>
#include <Console.h>

enum Data
{
	PARTICLE_X,
	PARTICLE_Y,
	PARTICLE_Z,
	PARTICLE_VX,
	PARTICLE_VY,
	PARTICLE_VZ,
	PARTICLE_LT,
	PARTICLE_LR
};

//Constants for particle
	//Life
	__constant__ float d_rLife[1], d_life[1];
	//Velocity
	__constant__ float d_initVelocityX[1], d_initVelocityY[1], d_initVelocityZ[1];
	__constant__ float d_rInitVelocityX[1], d_rInitVelocityY[1], d_rInitVelocityZ[1];
	__constant__ float d_vDecay[1];

//Constants for emitter
__constant__ unsigned int d_maxParticles[1], d_emitterFrec[1];


//Constans for wind
__constant__ float d_constantX[1], d_constantY[1], d_constantZ[1];
__constant__ unsigned int d_gridSize[1], d_perlinSize[1];



__global__ void setupRandomParticle( hiprandState * state, unsigned long seed)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx<d_maxParticles[0])
    hiprand_init ( seed, idx, 0, &state[idx] );
} 


__global__ void kernelParticle(float *x, float *y, float *z,
							float *vx, float *vy, float *vz,
							float *lt, float *lr,
							hiprandState* state, double dt)
{

	// Get our global thread ID
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if (id < d_maxParticles[0])
	{
		/* hiprand works like rand - except that it takes a state as a parameter */
		if(lr[id]<0.f)
		{
			//Space to create a particle
			int r = hiprand(&state[id])%1000;
			if(r<d_emitterFrec[0])
			{
				x[id] = 0.f;
				y[id] = 0.f;
				z[id] = 0.f;

				r=hiprand(&state[id])%1000;
				vx[id] = d_initVelocityX[0] + d_rInitVelocityX[0]*2.f*((r/1000.f)-0.5f);

				r=hiprand(&state[id])%1000;
				vy[id] = d_initVelocityY[0] + d_rInitVelocityY[0]*2.f*((r/1000.f)-0.5f);

				r=hiprand(&state[id])%1000;
				vz[id] = d_initVelocityZ[0] + d_rInitVelocityZ[0]*2.f*((r/1000.f)-0.5f);

				r=hiprand(&state[id])%1000;
				lt[id] = 0.f;
				lr[id] = d_life[0] - d_life[0]*d_rLife[0] + 2*d_life[0]*d_rLife[0]*r/1000.f;
			}
		}else
		{
			//Velocity Decay
			vx[id] = vx[id] - vx[id]*d_vDecay[0]*dt;
			vy[id] = vy[id] - vy[id]*d_vDecay[0]*dt;
			vz[id] = vz[id] - vz[id]*d_vDecay[0]*dt;

			//Wind constant velocity
			vx[id] = vx[id] + d_constantX[0];
			vy[id] = vy[id] + d_constantY[0];
			vz[id] = vz[id] + d_constantZ[0];

			//Position addition
			x[id] += vx[id]*dt;
			y[id] += vy[id]*dt;
			z[id] += vz[id]*dt;

			//Life set
			lr[id] = lr[id] - dt;
			lt[id] = lt[id] + dt;

		}
	}
}

__global__ void setupRandomPerlin(hiprandState * state, unsigned long seed)
{
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int idz = blockIdx.z*blockDim.z+threadIdx.z;

	if(idx>d_gridSize[0] || idy>d_gridSize[0] || idz>d_gridSize[0])
		return;

	unsigned int id = idx + idy*d_gridSize[0] + idz*d_gridSize[0]*d_gridSize[0];

	hiprand_init ( seed, id, 0, &state[idx]);
}

__global__ void addRandomPerlin(float *d_perlin_x, float *d_perlin_y, float *d_perlin_z, hiprandState* state)
{
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int idz = blockIdx.z*blockDim.z+threadIdx.z;

	if(idx>d_gridSize[0] || idy>d_gridSize[0] || idz>d_gridSize[0])
		return;

	unsigned int id = idx + idy*d_gridSize[0] + idz*d_gridSize[0]*d_gridSize[0];


}

__global__ void kernelPerlin(float *d_perlin_x, float *d_perlin_y, float *d_perlin_z)
{
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int idz = blockIdx.z*blockDim.z+threadIdx.z;

	if(idx>d_gridSize[0] || idy>d_gridSize[0] || idz>d_gridSize[0])
		return;

	unsigned int id = idx + idy*d_gridSize[0] + idz*d_gridSize[0]*d_gridSize[0];


}

int CudaControler::testDevices()
{
	int devCount;
		hipGetDeviceCount(&devCount);
		if(devCount<1)
		{
			cPrint("Error: No devices found\n", 1);
			return 1;
		}
			cPrint("Devices found: " + std::to_string(devCount) + "\nDevice using: ", 1);
			hipDeviceProp_t devProp;
			hipGetDeviceProperties(&devProp, 0);
			cPrint(devProp.name, 1);
			cPrint("\n",1);
			cudaGLSetGLDevice(0);

			cPrint("  Device Properties:\n", 2);
			cPrint("    >Total mem: " + cString(devProp.totalGlobalMem/(1024*1024))+" Mb\n" ,2);
			cPrint("    >Multiprocessor count: " + cString(devProp.multiProcessorCount)+"\n" ,2);
			cPrint("    >Max thread per Multiprocessor: " + cString(devProp.maxThreadsPerMultiProcessor)+"\n", 2);
			cPrint("      >Total: " + cString(devProp.multiProcessorCount*devProp.maxThreadsPerMultiProcessor) + "\n", 2);


		return 0;
}

std::string CudaControler::getDevice()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	return devProp.name;
}

void CudaControler::start()
{

	// Size, in bytes, of Particles vector host
	size_t bytes = values::e_MaxParticles*sizeof(float);
	if(values::sys_Double)
		bytes = values::e_MaxParticles*sizeof(double);

	//Allocate memory for resource vector in host
	h_resource = (float*)malloc(bytes);


	// Size, in bytes, of each 3D perlin matrix in device
	bytes = values::g_Size*values::g_Size*values::g_Size*sizeof(float);
	if(values::sys_Double)
		bytes = values::g_Size*values::g_Size*values::g_Size*sizeof(double);

	//Allocate memory for perlin noise grids in device
	cudaSafeCall(hipMalloc(&d_perlin_x, bytes));
	cudaSafeCall(hipMalloc(&d_perlin_y, bytes));
	cudaSafeCall(hipMalloc(&d_perlin_z, bytes));

	// Number of threads in each block
	particles_blockSize = values::cu_BlockSize;
	perlin_blockSize = values::cu_BlockSize;

	// Number of blocks in grid
	if(values::sys_Double)
	{
		particles_gridSize = (int)ceil((double)values::e_MaxParticles/particles_blockSize);
		perlin_gridSize = (int)ceil((double)(values::g_Size*values::g_Size*values::g_Size)/perlin_blockSize);
	}else
	{
		particles_gridSize = (int)ceil((float)values::e_MaxParticles/particles_blockSize);
		perlin_gridSize = (int)ceil((float)(values::g_Size*values::g_Size*values::g_Size)/perlin_blockSize);	
	}

	//calculatePerlin();

}

void CudaControler::close()
{
		// Release host memory
		free(h_resource);

		cudaSafeCall(hipFree(d_perlin_x));
		cudaSafeCall(hipFree(d_perlin_y));
		cudaSafeCall(hipFree(d_perlin_z));
}

void CudaControler::step(double dt)
{
	//Copy constant data to device
	copyConstants();

	//Maping the OpenGL buffers for CUDA
	size_t bytes = values::e_MaxParticles*sizeof(float);

	cudaSafeCall( hipGraphicsMapResources(1, &resource_x) );
	cudaSafeCall( hipGraphicsMapResources(1, &resource_y) );
	cudaSafeCall( hipGraphicsMapResources(1, &resource_z) );	
	cudaSafeCall( hipGraphicsMapResources(1, &resource_vx) );
	cudaSafeCall( hipGraphicsMapResources(1, &resource_vy) );
	cudaSafeCall( hipGraphicsMapResources(1, &resource_vz) );
	cudaSafeCall( hipGraphicsMapResources(1, &resource_lt) );
	cudaSafeCall( hipGraphicsMapResources(1, &resource_lr) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_x_s, &bytes, resource_x) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_y_s, &bytes, resource_y) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_z_s, &bytes, resource_z) );	
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_vx_s, &bytes, resource_vx) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_vy_s, &bytes, resource_vy) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_vz_s, &bytes, resource_vz) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_lt_s, &bytes, resource_lt) );
	cudaSafeCall( hipGraphicsResourceGetMappedPointer((void **)&d_lr_s, &bytes, resource_lr) );

	//Execute Kernel
	//Random device States
	hiprandState* devStates;
	hipMalloc ( &devStates, values::e_MaxParticles*sizeof( hiprandState ) );
	
	setupRandomParticle<<<particles_gridSize, particles_blockSize>>> ( devStates, rand()%10000);

	kernelParticle<<<particles_gridSize, particles_blockSize>>>(d_x_s, d_y_s, d_z_s, d_vx_s, d_vy_s, d_vz_s, d_lt_s, d_lr_s, devStates, dt);
	
	hipFree(devStates);

	//printData(PARTICLE_X);
	//printData(PARTICLE_VX);
	//printData(PARTICLE_Y);
	//printData(PARTICLE_VY);
	//printData(PARTICLE_Z);
	//printData(PARTICLE_VZ);
	//printData(PARTICLE_LT);
	//printData(PARTICLE_LR);


	//Reset the buffers
	hipGraphicsUnmapResources(1, &resource_x);
	hipGraphicsUnmapResources(1, &resource_y);
	hipGraphicsUnmapResources(1, &resource_z);
	hipGraphicsUnmapResources(1, &resource_vx);
	hipGraphicsUnmapResources(1, &resource_vy);
	hipGraphicsUnmapResources(1, &resource_vz);
	hipGraphicsUnmapResources(1, &resource_lt);
	hipGraphicsUnmapResources(1, &resource_lr);
}

void CudaControler::conectBuffers(unsigned int bufferX,unsigned int bufferY, unsigned int bufferZ,
									unsigned int bufferVX,unsigned int bufferVY, unsigned int bufferVZ, 
									unsigned int bufferLT, unsigned int bufferLR)
{	
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_x, (GLuint)bufferX, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_y, (GLuint)bufferY, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_z, (GLuint)bufferZ, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_vx, (GLuint)bufferVX, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_vy, (GLuint)bufferVY, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_vz, (GLuint)bufferVZ, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_lt, (GLuint)bufferLT, hipGraphicsRegisterFlagsNone) );
	cudaSafeCall( hipGraphicsGLRegisterBuffer(&resource_lr, (GLuint)bufferLR, hipGraphicsRegisterFlagsNone) );

	/*Unecesary
	// Initialize vectors on host
	for (size_t i = 0; i< sizeof(h_resource)/sizeof(*h_resource); i++)
		h_resource[i] = -1.f;

	// Copy host vectors to device
	size_t bytes = values::e_MaxParticles*sizeof(float);
	hipMemcpy( d_lr_s, h_resource, bytes, hipMemcpyHostToDevice);
	*/

}


void CudaControler::cudaSafeCall(hipError_t err){
  if(hipSuccess != err) {
	  std::string m = hipGetErrorString(err);
	  cPrint("Error in CUDA: " + m + "\n", 1);
  }
}


void CudaControler::printData(Data d)
{
	size_t bytes = values::e_MaxParticles*sizeof(float);
	if(values::sys_Double)
		bytes = values::e_MaxParticles*sizeof(double);

	cPrint("Cuda:   ", 2);

	// Copy array back to host
	switch(d)
	{
		case PARTICLE_X:
			cPrint("X:   ", 2);
			hipMemcpy( h_resource, d_x_s, bytes, hipMemcpyDeviceToHost );
		break;		
		case PARTICLE_Y:
			cPrint("Y:   ", 2);
			hipMemcpy( h_resource, d_y_s, bytes, hipMemcpyDeviceToHost );
		break;
		case PARTICLE_Z:
			cPrint("Z:   ", 2);
			hipMemcpy( h_resource, d_z_s, bytes, hipMemcpyDeviceToHost );
		break;
		case PARTICLE_VX:
			cPrint("VX:   ", 2);
			hipMemcpy( h_resource, d_vx_s, bytes, hipMemcpyDeviceToHost );
		break;
		case PARTICLE_VY:
			cPrint("VY:   ", 2);
			hipMemcpy( h_resource, d_vy_s, bytes, hipMemcpyDeviceToHost );
		break;
		case PARTICLE_VZ:
			cPrint("VZ:   ", 2);
			hipMemcpy( h_resource, d_vz_s, bytes, hipMemcpyDeviceToHost );
		break;
		case PARTICLE_LT:
			cPrint("LT:   ", 2);
			hipMemcpy( h_resource, d_lt_s, bytes, hipMemcpyDeviceToHost );
		break;
		case PARTICLE_LR:
			cPrint("LR:   ", 2);
			hipMemcpy( h_resource, d_lr_s, bytes, hipMemcpyDeviceToHost );
		break;
		default:
			cPrint("X:   ", 2);
			hipMemcpy( h_resource, d_x_s, bytes, hipMemcpyDeviceToHost );
		break;
	}

	for(int i = 0; i<values::e_MaxParticles; i++)
	{
		cPrint(cString(h_resource[i]) + " ", 2);
	}
	cPrint("\n", 1);
}


void CudaControler::copyConstants()
{
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_rLife), 			&(values::p_RLifeTime),			sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_life), 			&(values::p_LifeTime), 			sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_vDecay), 			&(values::p_VelocityDecay), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_initVelocityX), 	&(values::p_InitVelocityX), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_initVelocityY), 	&(values::p_InitVelocityY), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_initVelocityZ), 	&(values::p_InitVelocityZ), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_rInitVelocityX),	&(values::p_RInitVelocityX), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_rInitVelocityY),	&(values::p_RInitVelocityY), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_rInitVelocityZ),	&(values::p_RInitVelocityZ), 	sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_maxParticles), 	&(values::e_MaxParticles), 		sizeof(const unsigned int)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_emitterFrec), 		&(values::e_EmissionFrec), 		sizeof(const unsigned int)));


	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_constantX),		&(values::w_ConstantX), 		sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_constantY),		&(values::w_ConstantY), 		sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_constantZ),		&(values::w_ConstantZ), 		sizeof(const float)));
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_gridSize),			&(values::g_Size), 				sizeof(const unsigned int)));

}


void CudaControler::calculatePerlin()
{
	hiprandState* devStates;
	hipMalloc ( &devStates, values::g_Size*values::g_Size*values::g_Size*sizeof( hiprandState ) );
	
	setupRandomPerlin<<<perlin_gridSize, perlin_blockSize>>> ( devStates, rand()%10000);

	addRandomPerlin<<<perlin_gridSize, perlin_blockSize>>>(d_perlin_x, d_perlin_y, d_perlin_z, devStates);

	kernelPerlin<<<perlin_gridSize, perlin_blockSize>>>(d_perlin_x, d_perlin_y, d_perlin_z);
	
	hipFree(devStates);
}